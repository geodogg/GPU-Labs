#include "hip/hip_runtime.h"
#include "functions.cuh"

// cuda library
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

__global__
void KernelAdd(int n, float * p_sum, float * x, float * y){

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n){
    p_sum[i] = x[i] + y[i];

  return;
}
